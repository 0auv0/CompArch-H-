//GPU版本的矩阵相乘
#include <iostream>
#include <random>
#include <hip/hip_runtime.h>

using namespace std;
#define BLOCK_SIZE 64

__global__ void gemm_blocks(float *A, float *B, float *C,int N){
    if ((blockIdx.y * blockDim.y + threadIdx.y) * N + blockIdx.x * blockDim.x + threadIdx.x >= N * N) {
        return;
    }

    const int begin_A = blockIdx.y * blockDim.y * N;
    const int end_A = begin_A + N - 1;
    const int step_A = blockDim.x;

    const int begin_B = blockIdx.x * blockDim.x;
    const int step_B = blockDim.y * N;

    float result_temp = 0.0f;
    int i_B = begin_B;
    for (int i_A=begin_A; i_A < end_A; i_A += step_A, i_B += step_B)  {
        // shared memory
        __shared__ float ASum[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float BSum[BLOCK_SIZE][BLOCK_SIZE];

        // copy data to shared memory
        ASum[threadIdx.y][threadIdx.x] = A[i_A + threadIdx.y * N + threadIdx.x];
        BSum[threadIdx.y][threadIdx.x] = B[i_B + threadIdx.y * N + threadIdx.x];

        __syncthreads();

        for (int i = 0; i < BLOCK_SIZE; i++) {
            result_temp += ASum[threadIdx.y][i] * BSum[i][threadIdx.x];
        }

        __syncthreads();
    }

    C[blockIdx.y * blockDim.y * N + begin_B + threadIdx.y * N + threadIdx.x] = result_temp;
}

void matrixMul(float *A, float *B, float *C, int N, int block_size){
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, N*N*sizeof(float));
    hipMalloc(&d_B, N*N*sizeof(float));
    hipMalloc(&d_C, N*N*sizeof(float));

    hipMemcpy(d_A, A, N*N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N*N*sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    // dim3 block(16, 16);
    // dim3 grid((N + block.x - 1) / block.x, (N + block.y - 1) / block.y);
    dim3 block(block_size, block_size);
    dim3 grid((N + block.x - 1) / block.x, (N + block.y - 1) / block.y);

    hipEventRecord(start,   0);
    gemm_blocks<<<grid, block>>>(d_A, d_B, d_C, N);
    hipEventRecord(end, 0);

    hipMemcpy(C, d_C, N*N*sizeof(float), hipMemcpyDeviceToHost);

    hipEventSynchronize(end);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, end);

    cout << "MM_BLOCKS GPU Time: " << milliseconds << "ms" << endl;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}



void gemm_verify(float *A, float *B, float *C, int N){
    float *temp = new float[N*N];
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            float sum = 0;
            for(int k = 0; k < N; k++){
                sum += A[i*N + k] * B[k*N + j];
            }
            temp[i*N + j] = sum;
        }
    }

    for(int i = 0; i < N*N; i++){
        if(fabs(temp[i] - C[i]) > 1e-3){
            cout << "Error at " << i << endl;
            cout << "temp: " << temp[i] << " C: " << C[i] << endl;
            cout << "Error: " << fabs(temp[i] - C[i]) << endl;
            delete[] temp;
            return;
        }
    }

    cout << "Correct" << endl;
    delete[] temp;
    return;
}

int main(int argc, char* argv[]){
  int seed, n, N, block_size;
    if(argc == 2){
        n = atoi(argv[1]);
        seed = time(NULL);
        block_size = 16;
    }
    else if(argc == 3){
        n = atoi(argv[1]);
        seed = atoi(argv[2]);
        block_size = 16;
    }
    else{
        n = atoi(argv[1]);
        seed = atoi(argv[2]);
        block_size = atoi(argv[3]);
    }

    N = 1 << n;
    float *A = new float[N*N];
    float *B = new float[N*N];
    float *C = new float[N*N];

    srand(seed);
    for(int i = 0; i < N*N; i++){
        A[i] = (float)rand() / RAND_MAX;
        B[i] = (float)rand() / RAND_MAX;
    }


    matrixMul(A, B, C, N, block_size);

    gemm_verify(A, B, C, N);

    delete[] A;
    delete[] B;
    delete[] C;

    return 0;
}

