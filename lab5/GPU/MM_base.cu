//GPU版本的矩阵相乘
#include <iostream>
#include <random>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void gemm_baseline(float *A, float *B, float *C,int n){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        float sum = 0.0f;
        for (int k = 0; k < n; ++k) {
            sum += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }

}

void matrixMul(float *A, float *B, float *C, int N, int block_size){
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, N*N*sizeof(float));
    hipMalloc(&d_B, N*N*sizeof(float));
    hipMalloc(&d_C, N*N*sizeof(float));

    hipMemcpy(d_A, A, N*N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N*N*sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    dim3 block(block_size, block_size);
    dim3 grid((N + block.x - 1) / block.x, (N + block.y - 1) / block.y);

    hipEventRecord(start, 0);
    gemm_baseline<<<grid, block>>>(d_A, d_B, d_C, N);
    hipEventRecord(end, 0);

    hipMemcpy(C, d_C, N*N*sizeof(float), hipMemcpyDeviceToHost);

    hipEventSynchronize(end);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, end);

    cout << "MM GPU Time: " << milliseconds << "ms" << endl;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}



void gemm_verify(float *A, float *B, float *C, int N){
    float *temp = new float[N*N];
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            float sum = 0;
            for(int k = 0; k < N; k++){
                sum += A[i*N + k] * B[k*N + j];
            }
            temp[i*N + j] = sum;
        }
    }

    for(int i = 0; i < N*N; i++){
        if(fabs(temp[i] - C[i]) > 1e-3){
            cout << "Error at " << i << endl;
            cout << "temp: " << temp[i] << " C: " << C[i] << endl;
            cout << "Error: " << fabs(temp[i] - C[i]) << endl;
            delete[] temp;
            return;
        }
    }

    cout << "Correct" << endl;
    delete[] temp;
    return;
}

int main(int argc, char* argv[]){
  int seed, n, N, block_size;
    if(argc == 2){
        n = atoi(argv[1]);
        seed = time(NULL);
        block_size = 16;
    }
    else if(argc >= 3){
        n = atoi(argv[1]);
        seed = atoi(argv[2]);
        block_size = 16;
    }
    else{
        n = atoi(argv[1]);
        seed = atoi(argv[2]);
        block_size = atoi(argv[3]);
    }

    N = 1 << n;
    float *A = new float[N*N];
    float *B = new float[N*N];
    float *C = new float[N*N];

    srand(seed);
    for(int i = 0; i < N*N; i++){
        A[i] = (float)rand() / RAND_MAX;
        B[i] = (float)rand() / RAND_MAX;
    }


    matrixMul(A, B, C, N, block_size);

    gemm_verify(A, B, C, N);

    delete[] A;
    delete[] B;
    delete[] C;

    return 0;
}
